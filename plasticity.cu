#include "hip/hip_runtime.h"
/* plasticity.cu
 *
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include "plasticity_kernel.cu"

#include "assist.h"

#define ERROR_CHECK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
    printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
    bool if_quiet = true;
    unsigned int timer_compute = 0;
    unsigned int timer_memory = 0;
    int i;
    char input_fn[1024];
    char output_fn[1024];
    data_type * deviceBetaP = NULL, *deviceSigma = NULL;
    data_type * deviceFlux = NULL, *deviceVel = NULL;
    int width = N, height = N;

    int seed = 0;
    CUT_DEVICE_INIT(argc, argv);
    cutGetCmdLineArgumenti(argc, (const char **) argv, "seed", &seed);

#ifdef LENGTHSCALE
    float lengthscale = 0.;
    cutGetCmdLineArgumentf(argc, (const char **) argv, "lengthscale", &lengthscale);
    printf("Lengthscale: %.2f\n", lengthscale);
#endif

#ifdef LOADING
    printf("Loading\n");
#else
    printf("Relaxing\n");
#endif
    printf("Running seed: %d\n", seed);

    if_quiet = true; // If not display matrix contents

    //printf("Input matrix file name: %s\n", input_fn);

    // -----------------------------------------------------------------------
    // Setup host side
    // -----------------------------------------------------------------------

    printf("Setup host side environment and launch kernel:\n");

    // allocate host memory for matrices M and N
    printf("  Allocate host memory for matrices.\n");
#ifdef DIMENSION3
    printf("    N: %d x %d x %d x %d\n", N, N, N, NUM_COMP);
    unsigned int size = N * N * N * NUM_COMP;
    int breadth = N;
#else
    printf("    N: %d x %d x %d\n", N, N, NUM_COMP);
    unsigned int size = N * N * NUM_COMP;
    int breadth = 1;
#endif
    unsigned int mem_size = sizeof(data_type) * size;
    data_type* hostBetaP = (data_type*) malloc(mem_size);
    data_type* hostSigma = (data_type*) malloc(mem_size);
    data_type* hostFlux = (data_type*) malloc(mem_size);
    data_type* hostVel = (data_type*) malloc(mem_size);

    // Initialize the input matrices.
    printf("  Initialize the input matrices.\n");

    double time = 0.;
#ifndef LENGTHSCALE
    sprintf(output_fn, FILE_PREFIX FILE_PREFIX2  "cuda_" RUN_DESC "_%d_" PRECISION_STR "_%d_L%d.plas", N, seed, lambda);
#else
    sprintf(output_fn, FILE_PREFIX FILE_PREFIX2  "cuda_" RUN_DESC "_%d_" PRECISION_STR "_%d_L%d_l%.2f.plas", N, seed, lambda, lengthscale);
#endif

#ifdef CONTINUE_RUN
    FILE *test_fp = fopen(output_fn, "rb");
    if (test_fp != NULL) {
        fclose(test_fp);
        test_fp=NULL;
        // Saved file exists
        // Load previous state 
        data_type * matrix;
        matrix = ReadMatrixFileFunc(output_fn, 1, breadth*height*width*NUM_COMP+1, 1, if_quiet);
        time = (double)*matrix;
        printf(" Restarting from t=%f\n", time);
        matrix++;
        for(i = 0; i < size; i++)
            hostBetaP[i] = (data_type) matrix[i];
        matrix--;
        free(matrix);
    } else 
    {
#endif
    // Load from relaxed or initialized file for runs
#ifdef LOADING
        data_type * matrix;
        sprintf(input_fn, FILE_PREFIX FILE_PREFIX2 "cuda_" RELAX_RUN_DESC "_%d_" PRECISION_STR "_%d_L%d.plas", N, seed, lambda);
        matrix = ReadMatrixFileFunc(input_fn, width, breadth*height*NUM_COMP, 1, if_quiet);
#else
        double * matrix;
        //float * matrix;
#ifndef LENGTHSCALE
        sprintf(input_fn, FILE_PREFIX FILE_PREFIX2 "initial_%d_%d.mat", N, seed);
#else
        sprintf(input_fn, FILE_PREFIX FILE_PREFIX2 "initial_%d_%d_L%.2f.mat", N, seed, lengthscale);
#endif
        matrix = ReadDoubleMatrixFile(input_fn, width, breadth*height*NUM_COMP, 0, if_quiet);
#endif
        for(i = 0; i < size; i++)
            hostBetaP[i] = (data_type) matrix[i];
        free(matrix); matrix = NULL;
    }

    double timeInc = 0.01;

#ifdef LOADING
    double endTime = 3.00/LOADING_RATE;
#else
    double endTime = 20.00;
#endif
    FILE *data_fp = OpenFile(output_fn, 
#ifdef CONTINUE_RUN
        "ab",
#else
        "wb", 
#endif
        if_quiet);
#define XSTR(s) STR(s)
#define STR(s) #s
    //FILE *data_fp = OpenFile("cudaload_"XSTR(N)"_dp_L%d.plas", "wb", if_quiet);

    // ===================================================================
    //  Allocate device memory for the input matrices.
    //  Copy memory from the host memory to the device memory.
    // ===================================================================

    CUT_SAFE_CALL(cutCreateTimer(&timer_memory));
    CUT_SAFE_CALL(cutStartTimer(timer_memory));

    printf("  Allocate device memory.\n");

    CUDA_SAFE_CALL(hipMalloc((void**) &deviceBetaP, mem_size));

    setupSystem();
    printf("  Copy host memory data to device.\n");
#ifdef DYNAMIC_NUCLEATION
    CUDA_SAFE_CALL(hipMalloc((void**) &beta0dot, mem_size));
    CUDA_SAFE_CALL(hipMemcpy(beta0dot, hostBetaP, mem_size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemset(deviceBetaP, 0, mem_size));

{    d_dim_vector L;
    L.x = width;
    L.y = height;
#ifdef DIMENSION3
    L.z = breadth;
#endif

    data_type *sigma;
    CUDA_SAFE_CALL(hipMalloc((void**) &sigma, mem_size)); 
    calculateSigma(beta0dot, sigma, L); 
    hipDeviceSynchronize();

    printf("max beta0 = %f\n", reduceMax(beta0dot, size));
    double max = 0.0; for (int i=0; i<size; i++){ if (hostBetaP[i] > max) max = hostBetaP[i];}
    printf("max host = %f\n", max);
    double sigmax = reduceMax(sigma, size);
    maxNucleationTimestep = 1.0/sqrt(fabs(sigmax));
    printf("sigmax  %f\n", sigmax);
    printf("maxNucleationTimestep = %f\n", maxNucleationTimestep);
    CUDA_SAFE_CALL(hipFree(sigma));
}
#else
    CUDA_SAFE_CALL(hipMemcpy(deviceBetaP, hostBetaP, mem_size,
        hipMemcpyHostToDevice));
#endif

    printf("  Allocate device memory for results.\n");

    // FIXME - These lines maybe allocating unused memory
    /*
    CUDA_SAFE_CALL(hipMalloc((void**) &deviceSigma, mem_size));
    hipMemset(deviceSigma, 0, mem_size);
    CUDA_SAFE_CALL(hipMalloc((void**) &deviceFlux, mem_size));
    hipMemset(deviceFlux, 0, mem_size);
    CUDA_SAFE_CALL(hipMalloc((void**) &deviceVel, mem_size));
    hipMemset(deviceVel, 0, mem_size);
    */

    CUT_SAFE_CALL(cutStopTimer(timer_memory));

    // ================================================
    // Initialize the block and grid dimensions here
    // ================================================

    printf("  Executing the kernel...\n");

    // Start the timer_compute to calculate how much time we spent on it.
    CUT_SAFE_CALL(cutCreateTimer(&timer_compute));
    CUT_SAFE_CALL(cutStartTimer(timer_compute));


    d_dim_vector L;
    L.x = width;
    L.y = height;
#ifdef DIMENSION3
    L.z = breadth;
#endif

    // If this is the initial slice
#ifndef LAST_SHOT_ONLY
    if (time==0.)
        ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*NUM_COMP, if_quiet); 
#endif

#ifndef DEBUG_TIMESTEPS
    while(time < endTime) {
        double intermediateTime;
#ifdef LOADING
        timeInc = 0.5;
#else
        if (time<=0.1) 
            timeInc = 0.01;
        else
            if (time <= 1.0)
                timeInc = 0.05;
            else
                if (time <= 5.0)
                    timeInc = 0.5;
                else
                    timeInc = 1.0;
#endif
        intermediateTime = time + timeInc;
        while(time < intermediateTime) {
            double timeStep = TVDstep(deviceBetaP, L, time, intermediateTime);
            printf("%le +%le\n", time, timeStep);
            time += timeStep;
        }
        hipDeviceSynchronize();
        hipMemcpy(hostBetaP, deviceBetaP, mem_size, hipMemcpyDeviceToHost);
#ifndef LAST_SHOT_ONLY
        ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*NUM_COMP, if_quiet); 
#endif
    }
#ifdef LAST_SHOT_ONLY
    ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*NUM_COMP, if_quiet);
#endif

#else
#ifndef SINGLE_STEP_DEBUG
    int count = 0;
    while(count++ < 10) {
        double intermediateTime = time+1.0;
        double timeStep = TVDstep(deviceBetaP, L, height, time, intermediateTime);
        printf("dbg %le +%le\n", time, timeStep);
        time += timeStep;

        hipDeviceSynchronize();
        hipMemcpy(hostBetaP, deviceBetaP, mem_size, hipMemcpyDeviceToHost);
        ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*NUM_COMP, if_quiet); 
    }
#else
#ifdef DIMENSION3
#error
#endif
    dim3 grid(N/TILEX, N);
    dim3 tids(TILEX, 3, 3);
    data_type *sigma;
    CUDA_SAFE_CALL(hipMalloc((void**) &sigma, sizeof(data_type)*breadth*width*height*NUM_SIG_COMP));
    data_type *rhs;
    CUDA_SAFE_CALL(hipMalloc((void**) &rhs, sizeof(data_type)*breadth*width*height*NUM_COMP));
    data_type *velocity;
    CUDA_SAFE_CALL(hipMalloc((void**) &velocity, sizeof(data_type)*breadth*width*height*NUM_COMP));

    calculateSigma(deviceBetaP, sigma, width, height);
    hipDeviceSynchronize();
    hipMemcpy(hostBetaP, sigma, mem_size, hipMemcpyDeviceToHost);
    ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*NUM_COMP, if_quiet); 

    // calculate flux
    centralHJ<<<grid, tids>>>(deviceBetaP, sigma, rhs, velocity, L);

    hipDeviceSynchronize();
    hipMemcpy(hostBetaP, rhs, mem_size, hipMemcpyDeviceToHost);
    ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*NUM_COMP, if_quiet); 
    
    hipMemcpy(hostBetaP, velocity, mem_size/NUM_COMP, hipMemcpyDeviceToHost);
    ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*NUM_COMP, if_quiet); 
#endif
#endif
    // Make sure all threads have finished their jobs
    // before we stop the timer_compute.
    hipDeviceSynchronize();

    fclose( data_fp );
    
    // Stop the timer_compute
    CUT_SAFE_CALL(cutStopTimer(timer_compute));

    // check if kernel execution generated an error
    ERROR_CHECK
    CUT_CHECK_ERROR("Kernel execution failed");

    // ===================================================================
    // Copy the results back from the host
    // ===================================================================

    printf("  Copy result from device to host.\n");

    CUT_SAFE_CALL(cutStartTimer(timer_memory));
    //hipMemcpy(hostSigma, deviceSigma, mem_size, hipMemcpyDeviceToHost);
    //hipMemcpy(hostFlux, deviceFlux, mem_size, hipMemcpyDeviceToHost);
    //hipMemcpy(hostVel, deviceVel, mem_size, hipMemcpyDeviceToHost);
    CUT_SAFE_CALL(cutStopTimer(timer_memory));

    // ================================================
    // Show timing information
    // ================================================

    printf("  GPU memory access time: %f (ms)\n",
        cutGetTimerValue(timer_memory));
    printf("  GPU computation time  : %f (ms)\n",
        cutGetTimerValue(timer_compute));
    printf("  GPU processing time   : %f (ms)\n",
        cutGetTimerValue(timer_compute) + cutGetTimerValue(timer_memory));
    CUT_SAFE_CALL(cutDeleteTimer(timer_memory));
    CUT_SAFE_CALL(cutDeleteTimer(timer_compute));

    //WriteMatrixFile("velocity.mat", hostVel, width, height, if_quiet);
    //WriteMatrixFile("rhs.mat", hostFlux, width, NUM_COMP*height, if_quiet); 
#if 0 
    for(i = 0; i < NUM_SIG_COMP; i++) {
        for(int j = 0; j < height; j++) {
            for(int k = 0; k < width; k++)
                fprintf(stdout, "%lf ", hostSigma[(i*height+j)*width+k]);
            fprintf(stdout, "\n");
        }
        fprintf(stdout, "\n");
    }
#endif
    // clean up memory
    free(hostBetaP); free(hostSigma);
    free(hostFlux); free(hostVel);

    // ===================================================================
    // Free the device memory
    // ===================================================================

    CUDA_SAFE_CALL(hipFree(deviceBetaP));
    //CUDA_SAFE_CALL(hipFree(deviceSigma));
    //CUDA_SAFE_CALL(hipFree(deviceFlux));
    //CUDA_SAFE_CALL(hipFree(deviceVel));
}


