/* plasticity.cu
 *
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include "plasticity_kernel.cu"

#include "assist.h"

#define ERROR_CHECK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
    printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
    bool if_quiet = true;
    unsigned int timer_compute = 0;
    unsigned int timer_memory = 0;
    int i;
    char input_fn[1024];
    char output_fn[1024];
    data_type * deviceBetaP = NULL, *deviceSigma = NULL;
    data_type * deviceFlux = NULL, *deviceVel = NULL;
    int width = N, height = N;

    int seed = 0;
    CUT_DEVICE_INIT(argc, argv);
    cutGetCmdLineArgumenti(argc, (const char **) argv, "seed", &seed);

#ifdef LOADING
    printf("Loading\n");
#else
    printf("Relaxing\n");
#endif
    printf("Running seed: %d\n", seed);

    if_quiet = true; // If not display matrix contents

    //printf("Input matrix file name: %s\n", input_fn);

    // -----------------------------------------------------------------------
    // Setup host side
    // -----------------------------------------------------------------------

    printf("Setup host side environment and launch kernel:\n");

    // allocate host memory for matrices M and N
    printf("  Allocate host memory for matrices.\n");
#ifdef DIMENSION3
    printf("    N: %d x %d x %d x 9\n", N, N, N);
    unsigned int size = N * N * N * 9;
    int breadth = N;
#else
    printf("    N: %d x %d x 9\n", N, N);
    unsigned int size = N * N * 9;
    int breadth = 1;
#endif
    unsigned int mem_size = sizeof(data_type) * size;
    data_type* hostBetaP = (data_type*) malloc(mem_size);
    data_type* hostSigma = (data_type*) malloc(mem_size);
    data_type* hostFlux = (data_type*) malloc(mem_size);
    data_type* hostVel = (data_type*) malloc(mem_size);

    // Initialize the input matrices.
    printf("  Initialize the input matrices.\n");

    double time = 0.;
    sprintf(output_fn, FILE_PREFIX "cuda_" RUN_DESC "_%d_" PRECISION_STR "_%d_L%d.plas", N, seed, lambda);

#ifdef CONTINUE_RUN
    FILE *test_fp = fopen(output_fn, "rb");
    if (test_fp != NULL) {
        fclose(test_fp);
        test_fp=NULL;
        // Saved file exists
        // Load previous state 
        data_type * matrix;
        matrix = ReadMatrixFileFunc(output_fn, 1, breadth*height*width*9+1, 1, if_quiet);
        time = (double)*matrix;
        printf(" Restarting from t=%f\n", time);
        matrix++;
        for(i = 0; i < size; i++)
            hostBetaP[i] = (data_type) matrix[i];
        matrix--;
        free(matrix);
    } else 
    {
#endif
    // Load from relaxed or initialized file for runs
#ifdef LOADING
        data_type * matrix;
        sprintf(input_fn, FILE_PREFIX "cuda_" RELAX_RUN_DESC "_%d_" PRECISION_STR "_%d_L%d.plas", N, seed, lambda);
        matrix = ReadMatrixFileFunc(input_fn, width, breadth*height*9, 1, if_quiet);
#else
        double * matrix;
        //float * matrix;
        sprintf(input_fn, FILE_PREFIX "initial_%d_%d.mat", N, seed);
        matrix = ReadDoubleMatrixFile(input_fn, width, breadth*height*9, 0, if_quiet);
        for(i = 0; i < size; i++)
            hostBetaP[i] = (data_type) matrix[i];
        free(matrix); matrix = NULL;
#endif
    }

    double timeInc = 0.01;

#ifdef LOADING
    double endTime = 4.00/LOADING_RATE;
#else
    double endTime = 20.00;
#endif
    FILE *data_fp = OpenFile(output_fn, 
#ifdef CONTINUE_RUN
        "ab",
#else
        "wb", 
#endif
        if_quiet);
#define XSTR(s) STR(s)
#define STR(s) #s
    //FILE *data_fp = OpenFile("cudaload_"XSTR(N)"_dp_L%d.plas", "wb", if_quiet);

    // ===================================================================
    //  Allocate device memory for the input matrices.
    //  Copy memory from the host memory to the device memory.
    // ===================================================================

    CUT_SAFE_CALL(cutCreateTimer(&timer_memory));
    CUT_SAFE_CALL(cutStartTimer(timer_memory));

    printf("  Allocate device memory.\n");

    CUDA_SAFE_CALL(hipMalloc((void**) &deviceBetaP, mem_size));

    printf("  Copy host memory data to device.\n");

    CUDA_SAFE_CALL(hipMemcpy(deviceBetaP, hostBetaP, mem_size,
        hipMemcpyHostToDevice));

    printf("  Allocate device memory for results.\n");

    CUDA_SAFE_CALL(hipMalloc((void**) &deviceSigma, mem_size));
    hipMemset(deviceSigma, 0, mem_size);
    CUDA_SAFE_CALL(hipMalloc((void**) &deviceFlux, mem_size));
    hipMemset(deviceFlux, 0, mem_size);
    CUDA_SAFE_CALL(hipMalloc((void**) &deviceVel, mem_size));
    hipMemset(deviceVel, 0, mem_size);

    CUT_SAFE_CALL(cutStopTimer(timer_memory));

    // ================================================
    // Initialize the block and grid dimensions here
    // ================================================

    printf("  Executing the kernel...\n");

    // Start the timer_compute to calculate how much time we spent on it.
    CUT_SAFE_CALL(cutCreateTimer(&timer_compute));
    CUT_SAFE_CALL(cutStartTimer(timer_compute));

    setupSystem();

    d_dim_vector L;
    L.x = width;
    L.y = height;
#ifdef DIMENSION3
    L.z = breadth;
#endif

    // If this is the initial slice
    if (time==0.)
        ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*9, if_quiet); 

#ifndef DEBUG_TIMESTEPS
    while(time < endTime) {
        double intermediateTime;
#ifdef LOADING
        timeInc = 0.5;
#else
        if (time<=0.1) 
            timeInc = 0.01;
        else
            if (time <= 1.0)
                timeInc = 0.05;
            else
                if (time <= 5.0)
                    timeInc = 0.5;
                else
                    timeInc = 1.0;
#endif
        intermediateTime = time + timeInc;
        while(time < intermediateTime) {
            double timeStep = TVD3rd(deviceBetaP, L, time, intermediateTime);
            printf("%le +%le\n", time, timeStep);
            time += timeStep;
        }
        hipDeviceSynchronize();
        hipMemcpy(hostBetaP, deviceBetaP, mem_size, hipMemcpyDeviceToHost);
        ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*9, if_quiet); 
    }
#else
#ifndef SINGLE_STEP_DEBUG
    int count = 0;
    while(count++ < 10) {
        double intermediateTime = time+1.0;
        double timeStep = TVD3rd(deviceBetaP, L, height, time, intermediateTime);
        printf("dbg %le +%le\n", time, timeStep);
        time += timeStep;

        hipDeviceSynchronize();
        hipMemcpy(hostBetaP, deviceBetaP, mem_size, hipMemcpyDeviceToHost);
        ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*9, if_quiet); 
    }
#else
#ifdef DIMENSION3
#error
#endif
    dim3 grid(N/TILEX, N);
    dim3 tids(TILEX, 3, 3);
    data_type *sigma;
    CUDA_SAFE_CALL(hipMalloc((void**) &sigma, sizeof(data_type)*breadth*width*height*9));
    data_type *rhs;
    CUDA_SAFE_CALL(hipMalloc((void**) &rhs, sizeof(data_type)*breadth*width*height*9));
    data_type *velocity;
    CUDA_SAFE_CALL(hipMalloc((void**) &velocity, sizeof(data_type)*breadth*width*height*9));

    calculateSigma(deviceBetaP, sigma, width, height);
    hipDeviceSynchronize();
    hipMemcpy(hostBetaP, sigma, mem_size, hipMemcpyDeviceToHost);
    ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*9, if_quiet); 

    // calculate flux
    centralHJ<<<grid, tids>>>(deviceBetaP, sigma, rhs, velocity, L);

    hipDeviceSynchronize();
    hipMemcpy(hostBetaP, rhs, mem_size, hipMemcpyDeviceToHost);
    ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*9, if_quiet); 
    
    hipMemcpy(hostBetaP, velocity, mem_size/9, hipMemcpyDeviceToHost);
    ContinueWriteMatrix( data_fp, hostBetaP, time, width, breadth*height*9, if_quiet); 
#endif
#endif
    // Make sure all threads have finished their jobs
    // before we stop the timer_compute.
    hipDeviceSynchronize();

    fclose( data_fp );
    
    // Stop the timer_compute
    CUT_SAFE_CALL(cutStopTimer(timer_compute));

    // check if kernel execution generated an error
    ERROR_CHECK
    CUT_CHECK_ERROR("Kernel execution failed");

    // ===================================================================
    // Copy the results back from the host
    // ===================================================================

    printf("  Copy result from device to host.\n");

    CUT_SAFE_CALL(cutStartTimer(timer_memory));
    hipMemcpy(hostSigma, deviceSigma, mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(hostFlux, deviceFlux, mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(hostVel, deviceVel, mem_size, hipMemcpyDeviceToHost);
    CUT_SAFE_CALL(cutStopTimer(timer_memory));

    // ================================================
    // Show timing information
    // ================================================

    printf("  GPU memory access time: %f (ms)\n",
        cutGetTimerValue(timer_memory));
    printf("  GPU computation time  : %f (ms)\n",
        cutGetTimerValue(timer_compute));
    printf("  GPU processing time   : %f (ms)\n",
        cutGetTimerValue(timer_compute) + cutGetTimerValue(timer_memory));
    CUT_SAFE_CALL(cutDeleteTimer(timer_memory));
    CUT_SAFE_CALL(cutDeleteTimer(timer_compute));

    //WriteMatrixFile("velocity.mat", hostVel, width, height, if_quiet);
    //WriteMatrixFile("rhs.mat", hostFlux, width, 9*height, if_quiet); 
#if 0 
    for(i = 0; i < 9; i++) {
        for(int j = 0; j < height; j++) {
            for(int k = 0; k < width; k++)
                fprintf(stdout, "%lf ", hostSigma[(i*height+j)*width+k]);
            fprintf(stdout, "\n");
        }
        fprintf(stdout, "\n");
    }
#endif
    // clean up memory
    free(hostBetaP); free(hostSigma);
    free(hostFlux); free(hostVel);

    // ===================================================================
    // Free the device memory
    // ===================================================================

    CUDA_SAFE_CALL(hipFree(deviceBetaP));
    CUDA_SAFE_CALL(hipFree(deviceSigma));
    CUDA_SAFE_CALL(hipFree(deviceFlux));
    CUDA_SAFE_CALL(hipFree(deviceVel));
}


