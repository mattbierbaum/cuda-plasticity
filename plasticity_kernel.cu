#include "hip/hip_runtime.h"
/* Plasticity.cu
 * This file contains the necessary kernels for execution
 *
 */

#ifndef _PLASTICITY_KERNEL_H_
#define _PLASTICITY_KERNEL_H_

#include <stdio.h>
#include "plasticity.h"

#include <cutil.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

#ifdef PYTHON_COMPATIBILITY_TRANSPOSE_FFT
#include "transpose_kernel.cu"
#endif

#ifdef PYTHON_COMPATIBILITY_FFTW
#include "fftw3.h"
#endif

#define Dx (1./(data_type)N)
#define TILEX 16

static hipfftHandle g_planr2c;
static hipfftHandle g_planc2r;

__device__ __inline__ data_type minmod3(data_type a, data_type b, data_type c)
{
    if (a>0 && b>0 && c>0) {
        return (a<b) ? ((a<c) ? a : c) : ((c<b) ? c : b);
    } else {
        if (a<0 && b<0 && c<0) {
            return (a>b) ? ((a>c) ? a : c) : ((c>b) ? c : b);
        } else
            return 0;
    }
}

__device__ __inline__ void
findDerivatives( data_type* u, int i, int j, d_dim_vector x, int coord, data_type *deriv_p, data_type *deriv_m, d_dim_vector L)
{
    volatile d_dim_vector d;
    d.x = (coord==0);
    d.y = (coord==1);
#ifdef DIMENSION3
    d.z = (coord==2);
#endif
    volatile int idx = i*3+j;
    volatile data_type diff_p, diff_m;
    volatile data_type val, val_l, val_r;
    val = locate(u, x, idx);
    val_r = locateop(u, x,+,d, idx);
    diff_p = val_r-val;
    val_l = locateop(u, x,-,d, idx);
    diff_m = val-val_l ;
//diff_p = (*(u+((idx)*L.y+((y+dy+L.y)%L.y))*L.x+((x+dx+L.x)%L.x)))- (*(u+((idx)*L.y+((y+L.y)%L.y))*L.x+((x+L.x)%L.x)));
//diff_m = (*(u+((idx)*L.y+((y+L.y)%L.y))*L.x+((x+L.x)%L.x)))- (*(u+((idx)*L.y+((y-dy+L.y)%L.y))*L.x+((x-dx+L.x)%L.x)));

#ifdef theta
    data_type val_rr, val_ll;
    data_type diff_1, diff_2, diff_3;
    val_rr = locateop(u, x,+2*,d, idx);
    diff_1 = theta*(val_rr + val - 2*val_r);
    diff_2 = theta*(val_r + val_l - 2*val);
    diff_3 = 0.5*(val_rr-val_r-val+val_l);
    *deriv_p = (diff_p-0.5*minmod3(diff_1,diff_2,diff_3)) * (data_type)N;

    val_ll = locateop(u, x,-2*,d, idx);
    diff_1 = theta*(val_ll + val - 2*val_l);
    diff_3 = 0.5*(val_r-val-val_l+val_ll);
    *deriv_m = (diff_m+0.5*minmod3(diff_1,diff_2,diff_3)) * (data_type)N;
#else
    /* LLF */
    *deriv_p = diff_p * (data_type)N;
    *deriv_m = diff_m * (data_type)N;
#endif
}

__device__ const int sigma_index[3][3] = {{0,1,2},{1,3,4},{2,4,5}};

__global__ void
centralHJ( data_type* u, data_type* sig, data_type* rhs, data_type* velocity, d_dim_vector L )
{
    volatile int bx = blockIdx.x;     
    int by = blockIdx.y;
#ifdef DIMENSION3
    int bz = blockIdx.z;
#endif
    /* x coordinate is split into threads */
    int tx = threadIdx.x;    
    /* Indices of the array this thread will tackle */
    int i = threadIdx.y;
    int j = threadIdx.z;
    int idx = i*3+j;
    int ix = bx*TILEX + tx;
#ifndef DIMENSION3
    int in_idx = by*L.x + ix;
#else
    int in_idx = (bz*L.y+by)*L.x+ix;
#endif
    volatile data_type derivative = 0.;
    volatile data_type ax, ay;
#ifdef DIMENSION3
    volatile data_type az;
#endif

#ifdef DIMENSION3
#define NUM_ELEM 6
#define NUM_ELEM2 8
#else
#define NUM_ELEM 4
#define NUM_ELEM2 4
#endif
    // these have all nine components 
    __shared__ data_type du[NUM_ELEM][3][3][TILEX];

    // these have only one among nine
    // we only need 6 for 3D, but make 8 for using for rhomod
    __shared__ data_type a[NUM_ELEM2][TILEX];
    
    // Specific to plasticity
    // Use a for rhomod, since it's only used in preparation
    //__shared__ data_type rhomod[4][TILEX];
    __shared__ data_type sigma[3][3][TILEX];
    __shared__ data_type v[NUM_ELEM2][3][TILEX];

    d_dim_vector x;
    x.x = ix;
    x.y = by;
#ifdef DIMENSION3
    x.z = bz;
#endif
    // Determine the derivatives
    findDerivatives(u, i, j, x, 0, &du[0][i][j][tx], &du[1][i][j][tx], L);
    findDerivatives(u, i, j, x, 1, &du[2][i][j][tx], &du[3][i][j][tx], L);
#ifdef DIMENSION3
    findDerivatives(u, i, j, x, 2, &du[4][i][j][tx], &du[5][i][j][tx], L);
#endif
    sigma[i][j][tx] = locate(sig, x, idx);
    __syncthreads();

#if 0
    // DEBUG
    if (idx<4)
        *(rhs+idx*L.y*L.x+in_idx) = du[idx][0][0][tx];
    __syncthreads();
    return;
    // DEBUG
#endif
    // checked up until this part. derivatives are correct.
#if 0
    // DEBUG
    *(rhs+idx*L.y*L.x+in_idx) = sigma[i][j][tx];
    __syncthreads();
    return;
    // DEBUG
#endif
    // checked up until this part. stresses are correct.

    // Prepare for calculation / Plasticity specific
    if (idx<NUM_ELEM2) {
        // Calculate rhomod
        volatile data_type rhomod = 0.;
        volatile int kx = idx%2;
#ifndef DIMENSION3
        volatile int ky = idx/2+2;
#else
        volatile int ky = (idx%4)/2+2;
        volatile int kz = idx/4+4;
#endif
        // rhomod += ux[i,j]*ux[i,j] when i!=x
        rhomod += du[kx][1][0][tx]*du[kx][1][0][tx];
        rhomod += du[kx][1][1][tx]*du[kx][1][1][tx];
        rhomod += du[kx][1][2][tx]*du[kx][1][2][tx];
        rhomod += du[kx][2][0][tx]*du[kx][2][0][tx];
        rhomod += du[kx][2][1][tx]*du[kx][2][1][tx];
        rhomod += du[kx][2][2][tx]*du[kx][2][2][tx];

        // rhomod += uy[i,j]*uy[i,j] when i!=y
        rhomod += du[ky][0][0][tx]*du[ky][0][0][tx];
        rhomod += du[ky][0][1][tx]*du[ky][0][1][tx];
        rhomod += du[ky][0][2][tx]*du[ky][0][2][tx];
        rhomod += du[ky][2][0][tx]*du[ky][2][0][tx];
        rhomod += du[ky][2][1][tx]*du[ky][2][1][tx];
        rhomod += du[ky][2][2][tx]*du[ky][2][2][tx];

#ifdef DIMENSION3
        // rhomod += uz[z,j]*uz[z,j] when i!=z
        rhomod += du[kz][0][0][tx]*du[kz][0][0][tx];
        rhomod += du[kz][0][1][tx]*du[kz][0][1][tx];
        rhomod += du[kz][0][2][tx]*du[kz][0][2][tx];
        rhomod += du[kz][1][0][tx]*du[kz][1][0][tx];
        rhomod += du[kz][1][1][tx]*du[kz][1][1][tx];
        rhomod += du[kz][1][2][tx]*du[kz][1][2][tx];
#endif

        // rhomod -= 2*ux[y,j]*uy[x,j]
        rhomod -= 2*du[kx][1][0][tx]*du[ky][0][0][tx];
        rhomod -= 2*du[kx][1][1][tx]*du[ky][0][1][tx];
        rhomod -= 2*du[kx][1][2][tx]*du[ky][0][2][tx];
#ifdef DIMENSION3
        // rhomod -= 2*ux[z,j]*uz[x,j]
        rhomod -= 2*du[kx][2][0][tx]*du[kz][0][0][tx];
        rhomod -= 2*du[kx][2][1][tx]*du[kz][0][1][tx];
        rhomod -= 2*du[kx][2][2][tx]*du[kz][0][2][tx];
        // rhomod -= 2*uy[z,j]*uz[y,j]
        rhomod -= 2*du[ky][2][0][tx]*du[kz][1][0][tx];
        rhomod -= 2*du[ky][2][1][tx]*du[kz][1][1][tx];
        rhomod -= 2*du[ky][2][2][tx]*du[kz][1][2][tx];
#endif

        if (rhomod < 0.)
            rhomod = 0.;

        // store in shm
        a[idx][tx] = sqrt(rhomod);
#ifndef DIMENSION3
    } else {
        if (idx<8) {
            volatile int tidx = idx-4;
#else
    } {
        if (idx<8) {
            volatile int tidx = idx;
#endif
            // Calculate velocity
            volatile data_type vx;
            volatile data_type vy;
            volatile data_type vz;
            volatile int kx = tidx%2;
#ifndef DIMENSION3
            volatile int ky = tidx/2+2;
#else
            volatile int ky = (tidx%4)/2+2;
            volatile int kz = tidx/4+4;
#endif

            // v[x] += uy[x][n]*sigma[y][n]
            // v[x] -= ux[y][n]*sigma[y][n]
            // v[x] -= ux[z][n]*sigma[z][n]
            // these two cancel
            // v[x] += ux[x][n]*sigma[x][n]
            // v[x] -= ux[x][n]*sigma[x][n]
            vx = du[ky][0][0][tx]*sigma[1][0][tx];
            vx -= du[kx][1][0][tx]*sigma[1][0][tx];
            vx -= du[kx][2][0][tx]*sigma[2][0][tx];
            vx += du[ky][0][1][tx]*sigma[1][1][tx];
            vx -= du[kx][1][1][tx]*sigma[1][1][tx];
            vx -= du[kx][2][1][tx]*sigma[2][1][tx];
            vx += du[ky][0][2][tx]*sigma[1][2][tx];
            vx -= du[kx][1][2][tx]*sigma[1][2][tx];
            vx -= du[kx][2][2][tx]*sigma[2][2][tx];
#ifdef DIMENSION3
            // v[x] += uz[x][n]*sigma[z][n]
            vx += du[kz][0][0][tx]*sigma[2][0][tx];
            vx += du[kz][0][1][tx]*sigma[2][1][tx];
            vx += du[kz][0][2][tx]*sigma[2][2][tx];
#endif
            // v[y] += ux[y][n]*sigma[x][n]
            // v[y] -= uy[x][n]*sigma[x][n]
            // v[y] -= uy[z][n]*sigma[z][n]
            vy = du[kx][1][0][tx]*sigma[0][0][tx];
            vy -= du[ky][0][0][tx]*sigma[0][0][tx];
            vy -= du[ky][2][0][tx]*sigma[2][0][tx];
            vy += du[kx][1][1][tx]*sigma[0][1][tx];
            vy -= du[ky][0][1][tx]*sigma[0][1][tx];
            vy -= du[ky][2][1][tx]*sigma[2][1][tx];
            vy += du[kx][1][2][tx]*sigma[0][2][tx];
            vy -= du[ky][0][2][tx]*sigma[0][2][tx];
            vy -= du[ky][2][2][tx]*sigma[2][2][tx];
#ifdef DIMENSION3
            // v[y] += uz[y][n]*sigma[z][n]
            vy += du[kz][1][0][tx]*sigma[2][0][tx];
            vy += du[kz][1][1][tx]*sigma[2][1][tx];
            vy += du[kz][1][2][tx]*sigma[2][2][tx];
#endif

            // v[z] += ux[z][n]*sigma[x][n]
            // v[z] += uy[z][n]*sigma[y][n]
            vz = du[kx][2][0][tx]*sigma[0][0][tx];
            vz += du[ky][2][0][tx]*sigma[1][0][tx];
            vz += du[kx][2][1][tx]*sigma[0][1][tx];
            vz += du[ky][2][1][tx]*sigma[1][1][tx];
            vz += du[kx][2][2][tx]*sigma[0][2][tx];
            vz += du[ky][2][2][tx]*sigma[1][2][tx];
#ifdef DIMENSION3
            // v[z] -= uz[x][n]*sigma[x][n]
            // v[z] -= uz[y][n]*sigma[y][n]
            // these two cancel
            // v[z] += uz[z][n]*sigma[z][n]
            // v[z] -= uz[z][n]*sigma[z][n]
            vz -= du[kz][0][0][tx]*sigma[0][0][tx];
            vz -= du[kz][0][1][tx]*sigma[0][1][tx];
            vz -= du[kz][0][2][tx]*sigma[0][2][tx];
            vz -= du[kz][1][0][tx]*sigma[2][0][tx];
            vz -= du[kz][1][1][tx]*sigma[2][1][tx];
            vz -= du[kz][1][2][tx]*sigma[2][2][tx];
#endif

            // FIXME - glide term
            data_type sigma_tr = (sigma[0][0][tx]+sigma[1][1][tx]+sigma[2][2][tx])/3.*lambda;
            vx += sigma_tr*(du[kx][0][0][tx]+du[kx][1][1][tx]+du[kx][2][2][tx]);
            vy += sigma_tr*(du[ky][0][0][tx]+du[ky][1][1][tx]+du[ky][2][2][tx]);
            vx -= sigma_tr*(du[kx][0][0][tx]+du[ky][0][1][tx]);
            vy -= sigma_tr*(du[kx][1][0][tx]+du[ky][1][1][tx]);
            vz -= sigma_tr*(du[kx][2][0][tx]+du[ky][2][1][tx]);
#ifdef DIMENSION3
            vz += sigma_tr*(du[kz][0][0][tx]+du[kz][1][1][tx]+du[kz][2][2][tx]);
            vx -= sigma_tr*(du[kz][0][2][tx]);
            vy -= sigma_tr*(du[kz][1][2][tx]);
            vz -= sigma_tr*(du[kz][2][2][tx]);
#endif
            // store in shm
            v[tidx][0][tx] = vx;
            v[tidx][1][tx] = vy;
            v[tidx][2][tx] = vz;
        }
    }
    __syncthreads();

    // FIXME - checked up until this part. rho and velocities are correct.

    if (idx < NUM_ELEM2) {
        volatile data_type irhomod = a[idx][tx];
#ifndef PYTHON_COMPATIBILITY_DIVIDE
#ifndef SLOPPY_NO_DIVIDE_BY_ZERO
        if (irhomod == 0.) {
            irhomod = 0.;
        } else {
            irhomod = 1./irhomod;
        } 
#else
        irhomod = 1./(irhomod+ME);
#endif
        // This gives different values - FIXME Check why
        v[idx][0][tx] *= irhomod;
        v[idx][1][tx] *= irhomod;
        v[idx][2][tx] *= irhomod;
        a[idx][tx] = 0.;
#else
        irhomod += sqrt(ME);
        v[idx][0][tx] /= irhomod;
        v[idx][1][tx] /= irhomod;
        v[idx][2][tx] /= irhomod;
        a[idx][tx] = 0.;
#endif
    }
    __syncthreads();
#if 0
    // DEBUG
    if (idx<4)
        *(rhs+(idx)*L.y*L.x+in_idx) = v[idx][0][tx];
    if (idx>=4 && idx<8)
        *(rhs+(idx)*L.y*L.x+in_idx) = v[idx-4][1][tx];
    __syncthreads();
    return;
    // DEBUG
#endif

    // Calculate the velocities 
    // NOTE: diverging branches for different i&j are not problems! 
    // because they are different warps (or the hope is so)
    // however one needs to be careful that they do not need to excute in order.
#ifdef LLF
    if (idx==0) {
        volatile data_type max = 0.;
        for(int k = 0; k<NUM_ELEM2; k++) {
            volatile data_type t = v[k][idx][tx];
            if (max < t) 
                max = t;
            if (max < -t)
                max = -t;
        }
        a[0][tx] = max;
        a[1][tx] = max;
        a[2][tx] = max;
        a[3][tx] = max;
#ifdef DIMENSION3
        a[4][tx] = max;
        a[5][tx] = max;
#endif
    }
#else
#ifndef DIMENSION3
    if (idx<2) 
#else
    if (idx<3)
#endif
    {
        // Calculate ab_pm
        for(int k = 0; k<NUM_ELEM2; k++) {
            volatile data_type t = v[k][idx][tx];
            if (a[idx*2][tx] < t) 
                a[idx*2][tx] = t;
            if (a[idx*2+1][tx] < -t)
                a[idx*2+1][tx] = -t;
        }
    } 
#endif
    __syncthreads();
#if 0
    // DEBUG
    if (idx<4)
        *(rhs+(idx)*L.y*L.x+in_idx) = a[idx][tx];
    if (idx>=4 && idx<8)
        *(rhs+(idx)*L.y*L.x+in_idx) = v[idx-4][0][tx];
    __syncthreads();
    return;
    // DEBUG
#endif

    // FIXME do this more elegantly
    ax = a[0][tx]+a[1][tx]+ME;
    ay = a[2][tx]+a[3][tx]+ME;
#ifdef DIMENSION3
    az = a[4][tx]+a[5][tx]+ME;
#endif

    // Calculate hamiltonian and derivative using these results
    for(int k=0; k<NUM_ELEM2; k++) {
        volatile data_type h;
        volatile int kx = k%2;
#ifndef DIMENSION3
        volatile int ky = k/2+2;
#else
        volatile int ky = (k%4)/2+2;
        volatile int kz = k/4+4;
#endif
        // H_ij = v_l (d_l beta_ij - d_i beta_lj)
        h = du[kx][i][j][tx] * v[k][0][tx] + du[ky][i][j][tx] * v[k][1][tx];
#ifdef DIMENSION3
        h += du[kz][i][j][tx] * v[k][2][tx];
#endif
        if (i==0) {
            h -= v[k][0][tx] * du[kx][0][j][tx];
            h -= v[k][1][tx] * du[kx][1][j][tx];
            h -= v[k][2][tx] * du[kx][2][j][tx];
        } else {
            if (i==1) {
                h -= v[k][0][tx] * du[ky][0][j][tx];
                h -= v[k][1][tx] * du[ky][1][j][tx];
                h -= v[k][2][tx] * du[ky][2][j][tx];
            }
#ifdef DIMENSION3
            else {
                if (i==2) {
                    h -= v[k][0][tx] * du[kz][0][j][tx];
                    h -= v[k][1][tx] * du[kz][1][j][tx];
                    h -= v[k][2][tx] * du[kz][2][j][tx];
                }
            }
#endif
        }
        // Checked up until this point
        // Glide only correction
        // H_ij += lambda v_l / 3 (d_l beta_kk - d_k beta_lk)
        if (i==j) {
            h -= (v[k][0][tx]*(du[kx][1][1][tx]+du[kx][2][2][tx]-du[ky][0][1][tx])
                    +v[k][1][tx]*(du[ky][0][0][tx]+du[ky][2][2][tx]-du[kx][1][0][tx])
                    -v[k][2][tx]*(du[kx][2][0][tx]+du[ky][2][1][tx])
#ifdef DIMENSION3
                    -v[k][0][tx]*(du[kz][0][2][tx])
                    -v[k][1][tx]*(du[kz][1][2][tx])
                    +v[k][2][tx]*(du[kz][0][0][tx]+du[kz][1][1][tx]) 
#endif
                )/3.*lambda;
        }    
        // add all to the derivative
        volatile int nkx = 1-kx;
        volatile int nky = 5-ky;
#ifndef DIMENSION3
        derivative += -h*a[nkx][tx]*a[nky][tx]/ax/ay;
#else
        volatile int nkz = 9-kz;
        derivative += -h*a[nkx][tx]*a[nky][tx]*a[nkz][tx]/ax/ay/az;
#endif
    }

    // diffusion term FIXME
    //if (i!=0)
        derivative += (du[0][i][j][tx]-du[1][i][j][tx])*a[0][tx]*a[1][tx]/ax;
    //if (i!=1)
        derivative += (du[2][i][j][tx]-du[3][i][j][tx])*a[2][tx]*a[3][tx]/ay;
#ifdef DIMENSION3
        derivative += (du[4][i][j][tx]-du[5][i][j][tx])*a[4][tx]*a[5][tx]/az;
#endif
    *(rhs+idx*Lsize(L)+in_idx) = derivative;
    if (idx==0)
        *(velocity+in_idx) = ax+ay
#ifdef DIMENSION3
                            +az
#endif
                             ;
}

__global__ void
calculateKSigma( cdata_type* Ku, d_dim_vector L )
{
    // This kernel calculates the sigma in k-space
    // Since K-space field is supposedly unnecessary afterwards,
    // it is overwritten
    int bx = blockIdx.x;     
    int by = blockIdx.y;
#ifdef DIMENSION3
    int bz = blockIdx.z;
#endif
    /* x coordinate is split into threads */
    int tx = threadIdx.x;    
    /* Indices of the array this thread will tackle */
    int i = threadIdx.y;
    int j = threadIdx.z;
    //int idx = i*3+j;
    int ix = bx*TILEX + tx;
#ifndef DIMENSION3
    int in_idx = by*L.x + ix;
#else
    int in_idx = (bz*L.y+by)*L.x+ix;
#endif

    // FIXME - k values need to be properly dealt with
#ifndef PYTHON_COMPATIBILITY_TRANSPOSE_FFT
    data_type kx = ix*2.*M_PI;
#else
    data_type kx = ((ix>N/2)?ix-N:ix)*2.*M_PI;
#endif
    data_type ky = ((by>N/2)?by-N:by)*2.*M_PI;
    data_type kSq = kx*kx + ky*ky;
#ifdef DIMENSION3
    data_type kz = ((bz>N/2)?bz-N:bz)*2.*M_PI;
    kSq += kz*kz;
#endif
    data_type kSqSq = kSq*kSq;
    data_type k[3];

#ifdef PYTHON_COMPATIBILITY
    kSq += ME;
    kSqSq += ME;
#endif

    __shared__ cdata_type Ku_shm[3][3][TILEX];
    __shared__ cdata_type Ksig_shm[3][3][TILEX];
 
    k[0] = kx;
    k[1] = ky;
#ifndef DIMENSION3
    k[2] = 0;
#else
    k[2] = kz;
#endif
    if (ix < L.x) {
        Ku_shm[i][j][tx] = *(Ku+in_idx+Lsize(L)*(i*3+j));
        Ksig_shm[i][j][tx] = init_cdata(0.,0.);
    }
    __syncthreads();

#ifndef DIMENSION3
    if (ix == 0 && by == 0) 
#else
    if (ix == 0 && by == 0 && bz == 0)
#endif
    {
        // This is the constant part
        Ksig_shm[i][j][tx].x = 0.;
        Ksig_shm[i][j][tx].y = 0.;
        // New Boundary condition d_i u_j = 0
#if 1
        if (i==j) {
            cdata_type betaE_trace;
            betaE_trace.x = (Ku_shm[0][0][tx].x+Ku_shm[1][1][tx].x+Ku_shm[2][2][tx].x);
            betaE_trace.y = (Ku_shm[0][0][tx].y+Ku_shm[1][1][tx].y+Ku_shm[2][2][tx].y);
            Ksig_shm[i][j][tx].x -= 2*mu*nu/(1-2*nu)*betaE_trace.x;
            Ksig_shm[i][j][tx].y -= 2*mu*nu/(1-2*nu)*betaE_trace.y;
        }
        Ksig_shm[i][j][tx].x -= mu*(Ku_shm[i][j][tx].x+Ku_shm[j][i][tx].x);
        Ksig_shm[i][j][tx].y -= mu*(Ku_shm[i][j][tx].y+Ku_shm[j][i][tx].y);
#endif
    } else {
        if (ix < L.x) {
            for(int m = 0; m < 3; m++)
                for(int n = 0; n < 3; n++) {
                    data_type M = ((2*mu*nu/(1-nu))*((k[m]*k[n]*(i==j)+k[i]*k[j]*(m==n))/kSq - (i==j)*(m==n)) - mu*((i==m)*(j==n)+(i==n)*(j==m)) - (2*mu/(1-nu))*k[i]*k[j]*k[m]*k[n]/kSqSq + mu*(k[i]*k[n]*(j==m)+k[i]*k[m]*(j==n)+k[j]*k[n]*(i==m)+k[j]*k[m]*(i==n))/kSq);
                    Ksig_shm[i][j][tx].x += M * Ku_shm[m][n][tx].x;
                    Ksig_shm[i][j][tx].y += M * Ku_shm[m][n][tx].y;
                }
        }
    }
    if (ix < L.x) {
        Ksig_shm[i][j][tx].x /= N*N;
        Ksig_shm[i][j][tx].y /= N*N;
        *(Ku+in_idx+Lsize(L)*(i*3+j)) = Ksig_shm[i][j][tx];
        // Division for Normalization
    }
    __syncthreads();
}

#ifdef PYTHON_COMPATIBILITY_FFTW
fftw_plan plan_r2c;
fftw_plan plan_c2r;
#endif

void
setupSystem() {
#ifndef DIMENSION3
    hipfftPlan2d(&g_planr2c, N, N, FORWARD_FFT);
    hipfftPlan2d(&g_planc2r, N, N, BACKWARD_FFT);
#else
    hipfftPlan3d(&g_planr2c, N, N, N, FORWARD_FFT);
    hipfftPlan3d(&g_planc2r, N, N, N, BACKWARD_FFT);
#endif
    cublasInit();
#ifdef PYTHON_COMPATIBILITY_FFTW
    double *in = (double *)fftw_malloc(N*N*sizeof(double)); 
    fftw_complex *out = (fftw_complex *)fftw_malloc(N*N*sizeof(fftw_complex)); 
    plan_r2c = fftw_plan_dft_r2c_2d(N, N, in, out, FFTW_ESTIMATE | FFTW_UNALIGNED);
    plan_c2r = fftw_plan_dft_c2r_2d(N, N, out, in, FFTW_ESTIMATE | FFTW_UNALIGNED);
    fftw_free(in);
    fftw_free(out);
#endif
}

__host__ void
calculateSigma( data_type* u, data_type* sigma, d_dim_vector L )
{
    dim3 grid(KGridSize(L));
    dim3 tids(TILEX, 3, 3);
    cdata_type *Ku;
    CUDA_SAFE_CALL(hipMalloc((void**) &Ku, sizeof(cdata_type)*LKsize(L)*9));

    // Fourier transform u
#ifndef PYTHON_COMPATIBILITY_TRANSPOSE_FFT
    for(int i = 0; i < 3; i++)
        for(int j = 0; j < 3; j++)
            fft_r2c(g_planr2c, (fft_dtype_r*)(u+Lsize(L)*(3*i+j)), (fft_dtype_c*)(Ku+LKsize(L)*(3*i+j)));
#else
#ifdef PYTHON_COMPATIBILITY_FFTW
    double *in = (double *)fftw_malloc(N*N*sizeof(double));
    fftw_complex *out = (fftw_complex *)fftw_malloc(N*N*sizeof(fftw_complex));
#endif
    data_type *in_transpose;
    hipMalloc((void**)&in_transpose, sizeof(data_type)*Lsize(L));
    cdata_type *out_transpose;
    hipMalloc((void**)&out_transpose, sizeof(cdata_type)*LKsize(L));
    for(int i = 0; i < 3; i++)
        for(int j = 0; j < 3; j++) {
            dim3 grid((L.x+BLOCK_DIM-1)/BLOCK_DIM, (L.y+BLOCK_DIM-1)/BLOCK_DIM);
            dim3 tids(BLOCK_DIM, BLOCK_DIM);
            dim3 kgrid(((N/2+1)+BLOCK_DIM-1)/BLOCK_DIM, (N+BLOCK_DIM-1)/BLOCK_DIM);
            dim3 ktids(BLOCK_DIM, BLOCK_DIM);
            transpose<data_type><<<grid, tids>>>(in_transpose, u+Lsize(L)*(3*i+j), L.x, L.y);
#ifdef PYTHON_COMPATIBILITY_FFTW
            hipMemcpy(in, in_transpose, Lsize(L)*sizeof(double), hipMemcpyDeviceToHost);
            fftw_execute_dft_r2c(plan_r2c, in, out);
            hipMemcpy(out_transpose, out, (L.y/2+1)*L.x*sizeof(fftw_complex), hipMemcpyHostToDevice);
#else
            fft_r2c(g_planr2c, (fft_dtype_r*)(in_transpose), (fft_dtype_c*)(out_transpose));
#endif
            transpose<cdata_type><<<kgrid, ktids>>>(Ku+LKsize(L)*(3*i+j), out_transpose, L.y/2+1, L.x);
        }
    hipFree(in_transpose);
    hipFree(out_transpose);
    hipDeviceSynchronize();
#endif
    
    // calculateKSigma
#ifndef PYTHON_COMPATIBILITY_TRANSPOSE_FFT
    d_dim_vector newL = L;
    newL.x = L.x/2+1;
    calculateKSigma<<<grid, tids>>>(Ku, newL);
#else
    dim3 ngrid((N+TILEX-1)/TILEX, N/2+1);
    dim3 ntids(TILEX, 3, 3);
    calculateKSigma<<<ngrid, ntids>>>(Ku, L.x, L.y/2+1);
#endif

    // inverse Fourier kSigma
#ifndef PYTHON_COMPATIBILITY_TRANSPOSE_FFT
    for(int i = 0; i < 3; i++)
        for(int j = 0; j < 3; j++)
            fft_c2r(g_planc2r, (fft_dtype_c*)(Ku+LKsize(L)*(3*i+j)), (fft_dtype_r*)(sigma+Lsize(L)*(3*i+j)));
#else
    hipDeviceSynchronize();
    hipMalloc((void**)&in_transpose, sizeof(data_type)*Lsize(L));
    hipMalloc((void**)&out_transpose, sizeof(cdata_type)*LKsize(L));
    for(int i = 0; i < 3; i++)
        for(int j = 0; j < 3; j++) {
            dim3 grid((L.x+BLOCK_DIM-1)/BLOCK_DIM, (L.y+BLOCK_DIM-1)/BLOCK_DIM);
            dim3 tids(BLOCK_DIM, BLOCK_DIM);
            dim3 kgrid((N+BLOCK_DIM-1)/BLOCK_DIM, ((N/2+1)+BLOCK_DIM-1)/BLOCK_DIM);
            dim3 ktids(BLOCK_DIM, BLOCK_DIM);
            transpose<cdata_type><<<kgrid,ktids>>>(out_transpose, Ku+(L.y/2+1)*L.x*(3*i+j), L.x, L.y/2+1);
#ifdef PYTHON_COMPATIBILITY_FFTW
            hipMemcpy(out, out_transpose, (L.x/2+1)*L.y*sizeof(fftw_complex), hipMemcpyDeviceToHost);
            fftw_execute_dft_c2r(plan_c2r, out, in);
            hipMemcpy(in_transpose, in, Lsize(L)*sizeof(double), hipMemcpyHostToDevice);
#else
            fft_c2r(g_planc2r, (fft_dtype_c*)(out_transpose), (fft_dtype_r*)(in_transpose));
#endif
            transpose<data_type><<<grid,tids>>>(sigma+Lsize(L)*(3*i+j),in_transpose, L.x, L.y);
        }
#ifdef PYTHON_COMPATIBILITY_FFTW
    fftw_free(in);
    fftw_free(out);
#endif
    hipFree(in_transpose);
    hipFree(out_transpose);
#endif
    CUDA_SAFE_CALL(hipFree(Ku));
}

#ifdef LOADING
__global__ void
loadSigma( data_type t, data_type* sigma, d_dim_vector L )
{
    int bx = blockIdx.x;     int by = blockIdx.y;
    /* x coordinate is split into threads */
    int tx = threadIdx.x;    
    /* Indices of the array this thread will tackle */
    int i = threadIdx.y;
    int j = threadIdx.z;
    //int idx = i*3+j;
    int ix = bx*TILEX + tx;
    int in_idx = by*L.x + ix;
     
    const data_type load[3][3] = LOAD_DEF;

    *(sigma+in_idx+(i*3+j)*Lsize(L)) += 2.*mu*load[i][j]*LOADING_RATE*t; 
    if (i==j)
        *(sigma+in_idx+(i*3+j)*Lsize(L)) += 2.*mu*nu/(1-2*nu)*(load[0][0]+load[1][1]+load[2][2])*LOADING_RATE*t; 
}
#endif

__host__ void
calculateFlux( data_type t, data_type* u, data_type* rhs, data_type* velocity, d_dim_vector L )
{
    dim3 grid(GridSize(L));
    dim3 tids(TILEX, 3, 3);
    data_type *sigma;
    CUDA_SAFE_CALL(hipMalloc((void**) &sigma, sizeof(data_type)*Lsize(L)*9));

    calculateSigma(u, sigma, L);
    hipDeviceSynchronize();

#ifdef LOADING
    loadSigma<<<grid, tids>>>(t, sigma, L);
    hipDeviceSynchronize();
#endif

    // calculate flux
    centralHJ<<<grid, tids>>>(u, sigma, rhs, velocity, L);

    hipDeviceSynchronize();

    CUDA_SAFE_CALL(hipFree(sigma));
}

__host__ data_type
reduceMax( data_type* u, int size )
{
    int idx = Iamax(size, u, 1);
    data_type maxVel = 0.;
    CUDA_SAFE_CALL(hipMemcpy(&maxVel, u+idx-1,  sizeof(data_type), hipMemcpyDeviceToHost));
    return maxVel;
}

__host__ void
updateField( data_type* u, data_type timeStep, data_type *rhs, int size)
{
    axpy(9*size, timeStep, rhs, 1, u, 1);
}

__host__ double
simpleTVD( data_type* u, d_dim_vector L, data_type time, data_type endTime)
{
    data_type *rhs, *velocity;
    double timestep = 0.;
    CUDA_SAFE_CALL(hipMalloc((void**) &rhs, sizeof(data_type)*Lsize(L)*9));
    CUDA_SAFE_CALL(hipMalloc((void**) &velocity, sizeof(data_type)*Lsize(L)));

    calculateFlux(time, u, rhs, velocity, L);
    timestep = CFLsafeFactor / reduceMax(velocity, Lsize(L)) / N;
    if (time+timestep > endTime)
        timestep = endTime - time + ME;

    updateField(u, timestep, rhs, Lsize(L));
    CUDA_SAFE_CALL(hipFree(rhs));
    CUDA_SAFE_CALL(hipFree(velocity));
    return timestep;
} 

__host__ double
TVD3rd( data_type* u, d_dim_vector L, data_type time, data_type endTime)
{
    data_type *rhs, *velocity;
    data_type *L0, *L1, *L2;
    data_type *F0, *F1, *F2;
    double timestep = 0.;
    const data_type alpha[3][3] = {{1.,0.,0.}, {3./4.,1./4.,0.}, {1./3.,0.,2./3.}};
    const data_type beta[3][3] = {{1.,0.,0.}, {0.,1./4.,0.}, {0.,0.,2./3.}};

    CUDA_SAFE_CALL(hipMalloc((void**) &F0, sizeof(data_type)*Lsize(L)*9));
    CUDA_SAFE_CALL(hipMalloc((void**) &F1, sizeof(data_type)*Lsize(L)*9));
    CUDA_SAFE_CALL(hipMalloc((void**) &F2, sizeof(data_type)*Lsize(L)*9));
    hipMemset(F0, 0, sizeof(data_type)*Lsize(L)*9);
    hipMemset(F1, 0, sizeof(data_type)*Lsize(L)*9);
    hipMemset(F2, 0, sizeof(data_type)*Lsize(L)*9);
    CUDA_SAFE_CALL(hipMalloc((void**) &L0, sizeof(data_type)*Lsize(L)*9));
    CUDA_SAFE_CALL(hipMalloc((void**) &L1, sizeof(data_type)*Lsize(L)*9));
    CUDA_SAFE_CALL(hipMalloc((void**) &L2, sizeof(data_type)*Lsize(L)*9));
    hipMemset(L0, 0, sizeof(data_type)*Lsize(L)*9);
    hipMemset(L1, 0, sizeof(data_type)*Lsize(L)*9);
    hipMemset(L2, 0, sizeof(data_type)*Lsize(L)*9);
    CUDA_SAFE_CALL(hipMalloc((void**) &rhs, sizeof(data_type)*Lsize(L)*9));
    CUDA_SAFE_CALL(hipMalloc((void**) &velocity, sizeof(data_type)*Lsize(L)));

    calculateFlux(time, u, rhs, velocity, L);
    timestep = CFLsafeFactor / reduceMax(velocity, Lsize(L)) / N;
    if (time+timestep > endTime)
        timestep = endTime - time + ME;

    updateField(L0, timestep, rhs, Lsize(L));
    updateField(F0, alpha[0][0], u, Lsize(L));
    updateField(F0, beta[0][0], L0, Lsize(L));

    calculateFlux(time, F0, rhs, velocity, L);

    updateField(L1, timestep, rhs, Lsize(L));
    updateField(F1, alpha[1][0], u, Lsize(L));
    updateField(F1, beta[1][0], L0, Lsize(L));
    updateField(F1, alpha[1][1], F0, Lsize(L));
    updateField(F1, beta[1][1], L1, Lsize(L));

    calculateFlux(time, F1, rhs, velocity, L);

    updateField(L2, timestep, rhs, Lsize(L));
    updateField(u, alpha[2][0]-1, u, Lsize(L));
    updateField(u, beta[2][0], L0, Lsize(L));
    updateField(u, alpha[2][1], F0, Lsize(L));
    updateField(u, beta[2][1], L1, Lsize(L));
    updateField(u, alpha[2][2], F1, Lsize(L));
    updateField(u, beta[2][2], L2, Lsize(L));

    //vec_copy(Lsize(L)*9, F2, 1, u, 1);
    CUDA_SAFE_CALL(hipFree(F0));
    CUDA_SAFE_CALL(hipFree(F1));
    CUDA_SAFE_CALL(hipFree(F2));
    CUDA_SAFE_CALL(hipFree(L0));
    CUDA_SAFE_CALL(hipFree(L1));
    CUDA_SAFE_CALL(hipFree(L2));
    CUDA_SAFE_CALL(hipFree(rhs));
    CUDA_SAFE_CALL(hipFree(velocity));
    return timestep;
} 

__host__ void
runTVDSimple( data_type* u, d_dim_vector L, double time, double endTime )
{
    while(time < endTime) {
        double timeStep = simpleTVD(u, L, time, endTime);
        printf("%f +%f\n", time, timeStep);
        time += timeStep;
    }
}

__host__ void
runTVD( data_type* u, d_dim_vector L, double time, double endTime )
{
    while(time < endTime) {
        double timeStep = TVD3rd(u, L, time, endTime);
        printf("%f +%f\n", time, timeStep);
        time += timeStep;
    }
}

#endif // #ifndef _PLASTICITY_KERNEL_H_
